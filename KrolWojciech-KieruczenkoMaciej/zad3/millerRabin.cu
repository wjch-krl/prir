#include "common.hu"

class MillerRabinPrimeChecker : public PrimeChecker
{
private:
    unsigned int numberOfIterations;
    
public:
    MillerRabinPrimeChecker(unsigned int numberOfIterations)
    {
        this->numberOfIterations = numberOfIterations;
    }
    
    /**
    Check if number is propapbly prime
    **/
    virtual bool checkNumber(unsigned int number)
    {
        // return for small or even values
        if ( number==2 || number==3 ) 
        {
            return true;
        }
        if ( number<=1 || number % 2 == 0)
        {
            return false;
        }
        // Rewrite number-1 (this is event number) as multiper*2^exponent
        int exponent = 0;
        unsigned int tmp = number-1;
        while(!(tmp & 1))
        {
            exponent++;
            tmp >>= 1;
        }
        
        unsigned int multiper = (number-1) / (1<<exponent);
        
        //Alocate device memory
        hiprandState* devStates;
        unsigned int* devResult;
        unsigned int result;
        CHK_OK( hipMalloc ( &devStates, numberOfIterations*sizeof( hiprandState ) ));
        CHK_OK( hipMalloc ( &devResult, sizeof(unsigned int) )); 
        //Set flag to 0
        CHK_OK( hipMemset ( devResult, 0, sizeof(unsigned int)));  
        //Init hiprand
        initCurand<<<(numberOfIterations+31)/32,32>>>(devStates,123,numberOfIterations);
        //Check for prime number
        checkPrimeMillerRabin<<<(numberOfIterations+31)/32,32>>>(devStates,number,multiper,numberOfIterations,exponent,devResult);
        //Copy result back to Host
        CHK_OK(hipMemcpy(&result, devResult, sizeof(unsigned int), hipMemcpyDeviceToHost));
        return result == 0;
    }

};