#include "common.hu"

class MillerRabinPrimeChecker : public PrimeChecker
{
private:
    unsigned int numberOfIterations;
    
public:
    MillerRabinPrimeChecker(unsigned int numberOfIterations)
    {
        this->numberOfIterations = numberOfIterations;
    }
    
    virtual bool checkNumber(unsigned int n)
    {
        // Must have ODD n greater than THREE
        if ( n==2 || n==3 ) return true;
        if ( n<=1 || n % 2 == 0) return false;
        
        // Write n-1 as d*2^s by factoring powers of 2 from n-1
        int s = 0;
        for ( unsigned int m = n-1; !(m & 1); ++s, m >>= 1 ); 
        
        unsigned int d = (n-1) / (1<<s);
        
        hiprandState* devStates;
        unsigned int* devResult;
        unsigned int result;
        CHK_OK( hipMalloc ( &devStates, numberOfIterations*sizeof( hiprandState ) ));
        CHK_OK( hipMalloc ( &devResult, sizeof(unsigned int) )); 
        CHK_OK( hipMemset( devResult, 0, sizeof(unsigned int)));  
            
        setupRandom<<<(numberOfIterations+31)/32,32>>>(devStates,123,numberOfIterations);
        checkPrimeMillerRabin<<<(numberOfIterations+31)/32,32>>>(devStates,n,d,numberOfIterations,s,devResult);
        
        CHK_OK(hipMemcpy(&result, devResult, sizeof(unsigned int), hipMemcpyDeviceToHost));
        // n is *probably* prime
        return result == 0;
    }

};